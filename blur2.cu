#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string>
#include <opencv2/opencv.hpp>

using namespace cv;
using namespace std;

/**
* image: Image to be blurred
* i: Pixel position y
* j: Pixel position x
* return average channel vector
**/

__device__
void get_average(const int &y, const int &x, int *image, int *result, const int &height, const int &width, const int &kernel){
    int blue, green, red, dy, dx, cont, limit;
    blue = green = red = cont = 0;
    
    // Limit of adjacent positions
    limit = kernel / 2;

    // Traverse all adjacents positions
    for( int i = -limit; i <= limit; ++i ){
        for( int j = -limit; j <= limit; ++j ){
            dy = y + i;
            dx = x + j;

            // Verify if point is inside of image
            if( 0 <= dy && dy < height && 0 <= dx && dx < width ){
                blue += image[dy * width * 3 + dx * 3 + 0];
                green += image[dy * width * 3 + dx * 3 + 1];
                red += image[dy * width * 3 + dx * 3 + 2];
                ++cont;
            }
        }
    }

    // Calculate average of each channel
    blue /= cont;
    green /= cont;
    red /= cont;
	
	result[y * width * 3 + x * 3 + 0] = blue;
	result[y * width * 3 + x * 3 + 1] = green;
	result[y * width * 3 + x * 3 + 2] = red;
}

__global__
void blur(int *image, int *result, int *height, int *width, int *kernel) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int n = *height * *width;
    // Calculate pixel average for every pixel   
    int i, j;
    for(int k = index; k < n; k += stride) {
    	i = k / *width;
    	j = k % *width;
    	get_average(i, j, image, result, *height, *width, *kernel);
    }
}

int main(int argc, char** argv){

	Mat image, result;
	
	int *height_d;
	int *width_d;
	int *kernel_d;
	
	int height;
	int width;
	int kernel;
	int num_threads;
	int num_blocks;

    hipMalloc(&height_d, sizeof(int));
    hipMalloc(&width_d, sizeof(int));
    hipMalloc(&kernel_d, sizeof(int));
  
    if (argc != 5){
        printf("Use: ./blurSecuencial.out <img_path> <num_kernel> <num_threads> <num_blocks>\n");
        return -1;
    }

    image = imread(argv[1], 1);

    if ( !image.data ){
        printf("Image couldn't be read. \n");
        return -1;
    }

    kernel = atoi(argv[2]);
    num_threads = atoi(argv[3]);
    num_blocks = atoi(argv[4]);

    height = image.size().height;
    width = image.size().width;
	
	hipMemcpy(kernel_d, &kernel, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(width_d, &width, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(height_d, &height, sizeof(int), hipMemcpyHostToDevice);

	int *image_mat_d;
	int *image_mat = (int *) malloc (height * width * 3 * sizeof(int));
	
	int *result_mat_d;
	int *result_mat = (int *) malloc (height * width * 3 * sizeof(int));

  	hipMalloc(&image_mat_d, sizeof(int) * height * width * 3);
    hipMalloc(&result_mat_d, sizeof(int) * height * width * 3);
	
    // Create result image with same dimensions
    result = Mat(height, width, CV_8UC3);
    
    for (int i = 0; i < height; ++i) {
    	for (int j = 0; j < width; ++j) {
    		for (int k = 0; k < 3; ++k) {
    			image_mat[i * width * 3 + j * 3 + k] = image.at<Vec3b>(i, j)[k];
    		}
    	}
    }
    
    hipMemcpy(image_mat_d, image_mat, sizeof(int) * height * width * 3, hipMemcpyHostToDevice);
    
//	int num_threads = 256;
//	int num_blocks = (height * width + num_threads - 1) / num_threads;

    blur<<<num_blocks, num_threads>>>(image_mat_d, result_mat_d, height_d, width_d, kernel_d);

	hipMemcpy(result_mat, result_mat_d, sizeof(int) * height * width * 3, hipMemcpyDeviceToHost);

	for (int i = 0; i < height; ++i) {
    	for (int j = 0; j < width; ++j) {
    		for (int k = 0; k < 3; ++k) {
    			result.at<Vec3b>(i, j)[k] = result_mat[i * width * 3 + j * 3 + k];
    		}
    	}
    }
    

    string file_name;
    file_name = "blurred.jpg";
    imwrite(file_name, result);
    
    free(image_mat);
    free(result_mat);
    return 0;
}