#include "hip/hip_runtime.h"

/**
 * Blur-effect
 */

 #include <stdio.h>
 #include <math.h>
 // For the CUDA runtime routines (prefixed with "cuda_")
 #include <hip/hip_runtime.h>

 #include <stdlib.h>
 #include <cstdint>
 #include <opencv2/opencv.hpp>
 
  using namespace cv;
  using namespace std;
  

// function aviable only on the device

  __device__ void aplyBlur(const int &x, const int &y, int &kernel, const int &w, const int &h, int *input, int *output){
    // collect the average data of neighbours 
    int blue,green,red;
    blue=green=red=0;
    int n=0;
    int pixel_pos;

    for(int i = x - (kernel/2); i < x+(kernel/2); i++)
    {    
        for (int j = y-(kernel/2); j < y+(kernel/2); j++)
        {
            //check if the point is in the image limits
            if(0<=i && i<w-1 && 0<=j && j<h-1){
                pixel_pos = (i*w*3)+(j*3);
                blue += input[pixel_pos+0];
                green += input[pixel_pos+1];
                red += input[pixel_pos+2];
                n++;
            }
        }
    }
    
    if(n!=0){
         //write the average on the output image
        output[pixel_pos+0]=blue/n;
        output[pixel_pos+1]=green/n;
        output[pixel_pos+2]=red/n;
    }
   
}

 /**
  * CUDA Kernel Device code
  * 
  */ 
 /*****************************************************************************/
 
 __global__ void blur(int *input,int *output, int *kernel, int *totalThreads, int *width, int *height)
 {   
     
    int tn = (blockDim.x * blockIdx.x) + threadIdx.x+1;
    
    int ini = (int)((int)*width/(int)*totalThreads)*(tn);
    int fin = (int)((int)*width/(int)*totalThreads)+ini;
    for (int i = ini; i < fin; i++)
    {
        for (int j = 0; j < (int)*height; j++)
        {
            aplyBlur(i,j,*kernel, *width,*height,input, output);
        }
    }
     
 }
 
 
 /******************************************************************************
  * Host main routine
  */
 int main(int argc, char **argv)
 {   
     // define variables
     int h_threads;
     int h_kernel;
     int h_width;
     int h_height;

     int *d_threads;
     int *d_kernel;
     int *d_width;
     int *d_height;

     Mat output;
     Mat input;
    // handle errors

    hipError_t error = hipSuccess;
     //read parameters
     if ( argc != 6 )
    {
        printf("usage: ./blur-effect <Image_Path> <Image_out_Path> <KERNEL> <THREADS X BLOCK> <BLOCKS>\n");
        return -1;
    }
    h_kernel = atoi(argv[3]);
    int threadsXblock = atoi(argv[4]);
    int blocks = atoi(argv[5]);
    h_threads = threadsXblock* blocks;
    String oFile = argv[2];

    //read the image and set width and height
    input = imread( argv[1], IMREAD_COLOR );
    if ( !input.data )
    {
        printf("No image data \n");
        return -1;
    }
    h_width = input.cols;
    h_height =input.rows;
    // define the output as a clone of input image
    output = input.clone();
    //imwrite( oFile, output ); // just for test

    printf(" Processing image %s \n width: %d  - Heigh : %d \n",argv[1],h_width,h_height);

    printf("Kernel : %d   Threads x Block: %d   # of Blocks: %d \n",h_kernel,threadsXblock,blocks);

    int *d_input;
    int *d_output;
    int *h_input;
    int *h_output;

     // malloc and hipMalloc
     error = hipMalloc(&d_height,sizeof(int));
     if (error != hipSuccess){
        fprintf(stderr, "Failed to allocate mem for d_height (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
     error = hipMalloc(&d_kernel,sizeof(int));
     if (error != hipSuccess){
        fprintf(stderr, "Failed to allocate mem for d_kernel (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
     error = hipMalloc(&d_width,sizeof(int));
     if (error != hipSuccess){
        fprintf(stderr, "Failed to allocate mem for d_width (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
     error = hipMalloc(&d_threads,sizeof(int));
     if (error != hipSuccess){
        fprintf(stderr, "Failed to allocate mem for d_threads (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

     error = hipMalloc(&d_input,h_width*h_height*sizeof(int)*3);
     if (error != hipSuccess){
        fprintf(stderr, "Failed to allocate mem for d_input (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
     error = hipMalloc(&d_output,h_width*h_height*sizeof(int)*3);
     if (error != hipSuccess){
        fprintf(stderr, "Failed to allocate mem for d_output (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }


     size_t size = h_width * h_height * 3 * sizeof(int);
     h_input = (int *)malloc(size);
     h_output = (int *)malloc(size);

     // set initial values
     Vec3b pixel;
     //pixel = input.at<Vec3b>(Point(i,j));
     printf("ORIGINAL IMAGE \n");
     for(int i=0;i<h_width;i++){
       for(int j=0;j<h_height;j++){
        
        h_input[(j*h_width*3)+(i*3)+0]= input.at<Vec3b>(i,j)[0];
        h_input[(j*h_width*3)+(i*3)+1]= input.at<Vec3b>(i,j)[1];
        h_input[(j*h_width*3)+(i*3)+2]= input.at<Vec3b>(i,j)[2];
       }
     }

     for(int t =0;t<20;t++){
         for (int k=0;k<20;k++){
            printf("[%d , %d , %d] ",h_input[(k*h_width*3)+(t*3)+0],h_input[(k*h_width*3)+(t*3)+1], h_input[(k*h_width*3)+(t*3)+2]);
         }
         printf("\n");
     }

     // MemCpy: host to device

     error = hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);
     if (error != hipSuccess){
        fprintf(stderr, "Failed to copy on device (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_kernel, &h_kernel, sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess){
        fprintf(stderr, "Failed to  to copy on device(error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_threads, &h_threads, sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess){
        fprintf(stderr, "Failed to  to copy on device (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_width, &h_width, sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess){
        fprintf(stderr, "Failed to  to copy on device (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_height, &h_height, sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess){
        fprintf(stderr, "Failed to  to copy on device (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    printf("CudaMemcpy host to device done.\n");

     // Launch kernel 
     
     blur<<<blocks,threadsXblock>>>(d_input,d_output, d_kernel, d_threads, d_width, d_height);

     //last error
     error = hipGetLastError();
    if (error != hipSuccess){
        fprintf(stderr, "Failed to launch blur (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

     // MemCpy: device to host
     error = hipMemcpy(h_output, d_output, sizeof(int)*h_width*h_height*3, hipMemcpyDeviceToHost);
     if (error != hipSuccess){
        fprintf(stderr, "Failed to  to copy from device (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    printf("MODIFIED IMAGE \n");

    for(int t =0;t<20;t++){
        for (int k=0;k<20;k++){
           printf("[%d , %d , %d] ",h_output[(k*h_width*3)+(t*3)+0],h_output[(k*h_width*3)+(t*3)+1], h_output[(k*h_width*3)+(t*3)+2]);
        }
        printf("\n");
    }
     for(int i=0;i<h_width;i++){
       for(int j=0;j<h_height;j++){
        pixel = Vec3b(h_output[(j*h_width*3)+(i*3)+0],h_output[(j*h_width*3)+(i*3)+1], h_output[(j*h_width*3)+(i*3)+2]);
        output.at<Vec3b>(Point(i,j))= pixel;
       }
     }

     // save data
     //imwrite( oFile, output );

     // free memory

     hipFree(d_height);
     hipFree(d_width);
     hipFree(d_output);
     hipFree(d_input);
     hipFree(d_kernel);
     hipFree(d_threads);

     free(h_input);
     free(h_output);

     return 0;
 }
 
 