#include "hip/hip_runtime.h"

/**
 * Blur-effect
 */

 #include <stdio.h>
 #include <math.h>
 // For the CUDA runtime routines (prefixed with "cuda_")
 #include <hip/hip_runtime.h>

 #include <stdlib.h>
 #include <cstdint>
 #include <opencv2/opencv.hpp>
 
  using namespace cv;
  using namespace std;
  

// function aviable only on the device

  __device__ void aplyBlur(int &x, int &y, int &kernel, int &w, int &h, int *input, int *output){
    // collect the average data of neighbours 
    int blue,green,red;
    blue=green=red=0;
    int n=0;
    int pixel_pos;

    for(int i = x - (kernel/2); i < x+(kernel/2); i++)
    {    
        for (int j = y-(kernel/2); j < y+(kernel/2); j++)
        {
            //check if the point is in the image limits
            if(0<=i && i<width-1 && 0<=j && j<height-1){
                pixel_pos = (i*w*3)+(j*3);
                blue += input[pixel_pos+0];
                green += input[pixel_pos+1];
                red += input[pixel_pos+2];
                n++;
            }
        }
    }
    
    if(n!=0){
         //write the average on the output image
        output[pixel_pos+0]=blue/n;
        output[pixel_pos+1]=green/n;
        output[pixel_pos+2]=red/n;
    }
   
}

 /**
  * CUDA Kernel Device code
  * 
  */ 
 /*****************************************************************************/
 
 __global__ void blur(int *input,int *output, int *kernel, int *totalThreads, int *width, int *height)
 {   
     
    int tn = (blockDim.x * blockIdx.x) + threadIdx.x;
    
    int ini = (int)(width/totalThreads)*(tn-1);
    int fin = (int)(width/totalThreads)+ini;
    for (int i = ini; i < fin; i++)
    {
        for (int j = 0; j < height; j++)
        {
            aplyBlur(i,j,*kernel, *width,*height,input, output);
        }
    }
     
 }
 
 
 /******************************************************************************
  * Host main routine
  */
 int main(int argc, char **argv)
 {   
     // define variables
     int h_threads, h_kernel,h_width,h_height;
     int *d_threads;
     int *d_kernel;
     int *d_width;
     int *d_height;

     Mat output;
     Mat input;

     //read parameters
     if ( argc != 5 )
    {
        printf("usage: ./blur-effect <Image_Path> <Image_out_Path> <KERNEL> <THREADS X BLOCK> <BLOCKS>\n");
        return -1;
    }
    h_kernel = atoi(argv[3]);
    int threadsXblock = atoi(argv[4]);
    int blocks = atoi(argv[4]);
    h_threads = threadsXblock* blocks;
    String oFile = argv[2];

    //read the image and set width and height
    input = imread( argv[1], IMREAD_COLOR );
    if ( !input.data )
    {
        printf("No image data \n");
        return -1;
    }
    width = input.cols;
    height =input.rows;
    // define the output as a clone of input image
    output = input.clone();

    int *d_input;
    int *d_output;
    int *h_input;
    int *h_output;

     // malloc and hipMalloc
     hipMalloc(d_height,sizeof(int));
     hipMalloc(d_kernel,sizeof(int));
     hipMalloc(d_width,sizeof(int));
     hipMalloc(d_threads,sizeof(int));

     hipMalloc(&d_input,width*height*sizeof(int)*3);
     hipMalloc(&d_output,width*height*sizeof(int)*3);
     
     malloc(&h_input,width*height*sizeof(int)*3);
     malloc(&h_output,width*height*sizeof(int)*3);

     // set initial values
     Vec3b pixel;

     for(int i=0;i<width;i++){
       for(int j=0;j<height;j++){
        pixel = input.at<Vec3b>(Point(i,j));
        h_input[(j*width*3)+(i*3)+0]= pixel.val(0);
        h_input[(j*width*3)+(i*3)+1]= pixel.val(1);
        h_input[(j*width*3)+(i*3)+2]= pixel.val(2);
       }
     }

     // MemCpy: host to device

     hipMemcpy(d_input, h_input, sizeof(int)*width*height*3, hipMemcpyHostToDevice);
     hipMemcpy(d_kernel, h_kernel, sizeof(int), hipMemcpyHostToDevice);
     hipMemcpy(d_threads, h_threads, sizeof(int), hipMemcpyHostToDevice);
     hipMemcpy(d_width, h_width, sizeof(int), hipMemcpyHostToDevice);
     hipMemcpy(d_height, h_height, sizeof(int), hipMemcpyHostToDevice);

     // define blocks 

     // Launch kernel 
     
     blur<<<blocks,threadsXblock>>>(d_input,d_output, d_kernel, d_threads, d_width, d_height);

     // MemCpy: device to host
     hipMemcpy(h_output, d_output, sizeof(int)*width*height*3, hipMemcpyDeviceToHost);

     for(int i=0;i<width;i++){
       for(int j=0;j<height;j++){
        
        pixel = Vec3b(h_output[(j*width*3)+(i*3)+0],h_output[(j*width*3)+(i*3)+1], h_output[(j*width*3)+(i*3)+2]);
        output.at<Vec3b>(Point(i,j))= pixel;
       }
     }

     // save data
     imwrite( oFile, output );

     // free memory

     hipFree(d_height);
     hipFree(d_width);
     hipFree(d_output);
     hipFree(d_input);
     hipFree(d_kernel);
     hipFree(d_threads);

     free(h_input);
     free(h_output);

     return 0;
 }
 
 